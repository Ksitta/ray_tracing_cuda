#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <float.h>
#include <fstream>
#include <sstream>
#include <string>
#include <hiprand/hiprand_kernel.h>
#include "vec3.cuh"
#include "ray.cuh"
#include "sphere.cuh"
#include "hitable_list.cuh"
#include "camera.cuh"
#include "material.cuh"
#include "common.cuh"
#include "moving_sphere.cuh"
#include "common.cuh"
#include "rect.cuh"
#include "revsurface.cuh"
#include "mesh.cuh"

// Matching the C++ code would recurse enough into color() calls that
// it was blowing up the stack, so we have to turn this into a
// limited-depth loop instead.  Later code in the book limits to a max
// depth of 50, so we adapt this a few chapters early on the GPU.
__device__ Vec3 color(const ray& r, Hitable **world, hiprandState *local_rand_state) {
    ray cur_ray = r;
    Vec3 cur_attenuation = Vec3(1.0,1.0,1.0);
    Vec3 cur_color = Vec3(0, 0, 0);
    for(int i = 0; i < 50; i++) {
        HitRecord rec;
        if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
            ray scattered;
            Vec3 attenuation;
            Vec3 emitted = rec.mat_ptr->emitted(rec.u, rec.v, rec.p);
            cur_color += cur_attenuation * emitted;
            if(rec.mat_ptr->scatter(cur_ray, rec, attenuation, scattered, local_rand_state)) {
                cur_attenuation *= attenuation;
                cur_ray = scattered;
            }
            else {
                return cur_color;
            }
        }
        else {
            cur_color += cur_attenuation * Vec3(0,0,0);
            return cur_color;
        }
    }
    return Vec3(0.0,0.0,0.0); // exceeded recursion
}

__global__ void rand_init(hiprandState *rand_state) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprand_init(1984, 0, 0, rand_state);
    }
}

__global__ void render_init(int max_x, int max_y, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x + i;
    // Original: Each thread gets same seed, a different sequence number, no offset
    // hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
    // BUGFIX, see Issue#2: Each thread gets different seed, same sequence for
    // performance improvement of about 2x!
    hiprand_init(1984+pixel_index, 0, 0, &rand_state[pixel_index]);
}

__global__ void render(Vec3 *fb, int max_x, int max_y, int ns, Camera **cam, Hitable **world, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x + i;
    hiprandState local_rand_state = rand_state[pixel_index];
    Vec3 col(0,0,0);
    for(int s=0; s < ns; s++) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u, v, &local_rand_state);
        col += color(r, world, &local_rand_state);
    }
    rand_state[pixel_index] = local_rand_state;
    col /= float(ns);
    col[0] = sqrt(col[0]);
    col[1] = sqrt(col[1]);
    col[2] = sqrt(col[2]);
    fb[pixel_index] = col;
}

#define RND (hiprand_uniform(&local_rand_state))
__device__ int object_num = 0;
const int max_num_hitables = 1000;
Hitable **d_list;
__device__ int texture_num = 0;
const int max_num_texture = 1000;
texture **d_texturelist;

__device__ void add_object(Hitable **d_list, Hitable *object) {
    d_list[object_num++] = object;
}

// __global__ void create_world(hitable **d_list, hitable **d_world, Camera **d_camera, int nx, int ny, hiprandState *rand_state) {
//     if (threadIdx.x == 0 && blockIdx.x == 0) {
//         hiprandState local_rand_state = *rand_state;
//         auto red   = new Lambertian(vec3(.65, .05, .05));
//         auto white = new Lambertian(vec3(.73, .73, .73));
//         auto green = new Lambertian(vec3(.12, .45, .15));
//         auto light = new DiffuseLight(vec3(15, 15, 15));

//         add_object(d_list, new YZRect(0, 555, 0, 555, 555, green));
//         add_object(d_list, new YZRect(0, 555, 0, 555, 0, red));
//         add_object(d_list, new XZRect(213, 343, 227, 332, 554, light));
//         add_object(d_list, new XZRect(0, 555, 0, 555, 0, white));
//         add_object(d_list, new XZRect(0, 555, 0, 555, 555, white));
//         add_object(d_list, new XYRect(0, 555, 0, 555, 555, white));

//         float sf = 300;
//         vec3 vase[] = {
//             vec3(0.27, 0, 0) * sf,
//             vec3(0.29, 0.1, 0) * sf,
//             vec3(0.33, 0.2, 0) * sf,
//             vec3(0.40, 0.4, 0) * sf,
//             vec3(0.36, 0.6, 0) * sf,
//             vec3(0.21, 0.72, 0) * sf,
//             vec3(0.3, 1, 0) * sf,
//         };
//         add_object(d_list,
//                     // new Sphere(vec3(70, 0, 55), 19.8, new Lambertian(vec3(117 / 255.f,190 / 255.f, 204 / 255.f)))
//                     new RevSurface(vec3(278, 0, 400), new Curve(vase, 7), new Lambertian(vec3(117 / 255.f,190 / 255.f, 204 / 255.f)))
//                     // new Cylinder(24, 60, vec3(70, -5, 55), new Lambertian(vec3(117 / 255.f,190 / 255.f, 204 / 255.f)))
//         );


//         *rand_state = local_rand_state;
//         *d_world  = new HittableList(d_list, object_num);

//         // float aspect_ratio = 1.0;
//         // int image_width = 600;
//         // int samples_per_pixel = 200;
//         vec3 background = vec3(0,0,0);
//         vec3 lookfrom = vec3(278, 278, -800);
//         vec3 lookat = vec3(278, 278, 0);
//         float vfov = 40.0;
//         float dist_to_focus = 1;
//         float aperture = 0;

//         *d_camera   = new Camera(lookfrom,
//                                  lookat,
//                                  vec3(0,1,0),
//                                  vfov,
//                                  float(nx)/float(ny),
//                                  aperture,
//                                  dist_to_focus);
//     }
// }

__global__ void add_mesh(Hitable **d_list, Triangle *triangles, int triangles_cnt){
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        // add_object(d_list, new Mesh(triangles, triangles_cnt, new Metal(vec3(0.999, 0.999, 0.999), 0)));
        add_object(d_list, new Mesh(triangles, triangles_cnt, new Lambertian(Vec3(220 / 255.f, 174 / 255.f, 185 / 255.f))));
    }
}

__global__ void create_world(Hitable **d_list, Hitable **d_world, Camera **d_camera, int nx, int ny, hiprandState *rand_state, texture **d_tex) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        hiprandState local_rand_state = *rand_state;

        add_object(d_list, new YZRect(0, 1000, 0, 1000, 1, new Lambertian(Vec3(117 / 255.f,190 / 255.f, 204 / 255.f))));
        add_object(d_list, new XYRect(0, 1000, 0, 1000, 0, new Lambertian(Vec3(220 / 255.f, 174 / 255.f, 185 / 255.f))));
        add_object(d_list, new XZRect(0, 1000, 0, 1000, 0, new Lambertian(Vec3(200 / 255.f, 150 / 255.f, 123 / 255.f))));
        add_object(d_list, new XZRect(0, 1000, 0, 1000, 81.6, new Lambertian(Vec3(200 / 255.f, 150 / 255.f, 123 / 255.f))));
        add_object(d_list, new Sphere(Vec3(80, 681.6 - 0.285, 115), 600, new DiffuseLight(Vec3(10, 10, 10))));

        add_object(d_list, new Sphere(Vec3(60, 13, 100.6), 13, new Metal(Vec3(0.999, 0.999, 0.999), 0)));
        add_object(d_list, new Sphere(Vec3(128, 15, 110), 15, new Dielectric(1.5)));

        float sf = 60;

        Vec3 vase[] = {
            Vec3(0.27, 0, 0) * sf,
            Vec3(0.29, 0.1, 0) * sf,
            Vec3(0.33, 0.2, 0) * sf,
            Vec3(0.40, 0.4, 0) * sf,
            Vec3(0.36, 0.6, 0) * sf,
            Vec3(0.21, 0.72, 0) * sf,
            Vec3(0.3, 1, 0) * sf,
        };
        add_object(d_list,
                    // new Sphere(vec3(70, 0, 55), 19.8, new Lambertian(vec3(117 / 255.f,190 / 255.f, 204 / 255.f)))
                    new RevSurface(Vec3(70, -5, 55), new BezierCurve(vase, 7), new Lambertian(d_tex[0]))
                    // new Cylinder(24, 60, vec3(70, -5, 55), new Lambertian(vec3(117 / 255.f,190 / 255.f, 204 / 255.f)))
        );

        // add_object(d_list, new Triangle(vec3(0, 21.213, 121.213), vec3(30, 21.213, 121.213), vec3(30, 42.426, 100), new Lambertian(vec3(220 / 255.f, 174 / 255.f, 185 / 255.f))));
        *rand_state = local_rand_state;
        *d_world  = new HittableList(d_list, object_num);

        Vec3 lookfrom(140, 52, 180.6f);
        Vec3 lookat(50, 30, 50);
        float dist_to_focus = 1;
        float aperture = 0;
        *d_camera   = new Camera(lookfrom,
                                 lookat,
                                 Vec3(0,1,0),
                                 45,
                                 float(nx)/float(ny),
                                 aperture,
                                 dist_to_focus);
    }
}

__global__ void free_world(Hitable **d_list, Hitable **d_world, Camera **d_camera) {
    for(int i=0; i < object_num; i++) {
        delete d_list[i];
    }
    delete *d_world;
    delete *d_camera;
}

__global__ void add_texture_to_list(texture **d_list, unsigned char *data, int width, int height) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        d_list[texture_num++] = new ImageTexture(data, width, height);
    }
}

void add_texture(const char* filename, texture **d_list){
    int components_per_pixel = 3;
    int height, width;
    unsigned char *data = stbi_load(
                filename, &width, &height, &components_per_pixel, components_per_pixel);
    unsigned char *d_data;
    if (!data) {
        std::cerr << "ERROR: Could not load Texture image file '" << filename << "'.\n";
        width = height = 0;
        exit(1);
    }

    checkCudaErrors(hipMalloc((void**)&d_data, width * height * components_per_pixel * sizeof(unsigned char)));
    checkCudaErrors(hipMemcpy(d_data, data, width * height * components_per_pixel * sizeof(unsigned char), hipMemcpyHostToDevice));
    free(data);
    add_texture_to_list<<<1, 1>>>(d_list, d_data, width, height);
}

int main() {
    int nx = 1920;
    int ny = 1080;
    int ns = 10000;
    int tx = 16;
    int ty = 16;

    std::cerr << "Rendering a " << nx << "x" << ny << " image with " << ns << " samples per pixel ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = nx*ny;
    size_t fb_size = num_pixels*sizeof(Vec3);

    // allocate FB
    Vec3 *fb;

    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    // allocate random state
    hiprandState *d_rand_state;
    checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels*sizeof(hiprandState)));
    hiprandState *d_rand_state2;
    checkCudaErrors(hipMalloc((void **)&d_rand_state2, 1*sizeof(hiprandState)));

    // we need that 2nd random state to be initialized for the world creation
    rand_init<<<1,1>>>(d_rand_state2);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // make our world of hitables & the Camera

    checkCudaErrors(hipMalloc((void **)&d_list, max_num_hitables * sizeof(Hitable *)));
    Hitable **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(Hitable *)));
    Camera **d_camera;
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(Camera *)));

    checkCudaErrors(hipMalloc((void **)&d_texturelist, max_num_texture * sizeof(texture *)));

    // Add object here
    Triangle *triangles;
    int triangle_num;
    read_mesh("../mesh/cube.obj", &triangles, &triangle_num);
    add_mesh<<<1, 1>>>(d_list, triangles, triangle_num);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    add_texture("../imgs/2.jpg", d_texturelist);
    create_world<<<1,1>>>(d_list, d_world, d_camera, nx, ny, d_rand_state2, d_texturelist);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    clock_t start, stop;
    start = clock();
    // Render our buffer
    dim3 blocks(nx/tx+1,ny/ty+1);
    dim3 threads(tx,ty);
    render_init<<<blocks, threads>>>(nx, ny, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());


    render<<<blocks, threads>>>(fb, nx, ny,  ns, d_camera, d_world, d_rand_state);
    // checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    float timer_seconds = ((float)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    // Output FB as Image
    FILE *f = fopen("image.ppm", "w");

    fprintf(f, "P3\n%d %d\n%d\n", nx, ny, 255);

    for (int j = ny-1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j*nx + i;
            int ir = int(255.99f*fb[pixel_index].r());
            int ig = int(255.99f*fb[pixel_index].g());
            int ib = int(255.99f*fb[pixel_index].b());
            if(ir > 255){
                ir = 255;
            }
            if(ig > 255){
                ig = 255;
            }
            if(ib > 255){
                ib = 255;
            }
            fprintf(f, "%d %d %d ", ir, ig, ib);
        }
    }

    fclose(f);

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1,1>>>(d_list,d_world,d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(d_rand_state2));
    checkCudaErrors(hipFree(fb));

    hipDeviceReset();
}
